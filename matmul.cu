#define UTILS_IMPLEMENTATION
#include "utils.h"

#ifndef M
#define M 10
#define K 10
#define N 10
#endif

// MATMUL KERNEL
// C = A x B
//
// Dimensions
// A = (M, K)
// B = (K, N)
// C = (M, N)

int main() {
  int size_a = M * N;
  int size_b = K * N;
  int size_c = M * N;

  // allocate memory on the host
  float *A_h = rand_init(size_a);
  float *B_h = rand_init(size_b);
  float *C_h = malloc(size_c * sizeof(float));

  // allocate memory on the device
  float *A_d, *B_d, *C_d;
  check_err(hipMalloc(&A_d, size_a * sizeof(float)));
  check_err(hipMalloc(&B_d, size_b * sizeof(float)));
  check_err(hipMalloc(&C_d, size_c * sizeof(float)));

  // move the A and B matrice to device
  check_err(hipMemcpy(A_d, A_h, size_a * sizeof(float), hipMemcpyHostToDevice));
  check_err(hipMemcpy(B_d, B_h, size_b * sizeof(float), hipMemcpyHostToDevice));
}
