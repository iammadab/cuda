#include "hip/hip_runtime.h"
#define UTILS_IMPLEMENTATION
#include "utils.h"

#ifndef M
#define M 10
#define K 10
#define N 10
#endif

// MATMUL KERNEL
// C = A x B
//
// Dimensions (row, col)
// A = (M, K)
// B = (K, N)
// C = (M, N)

// TODO: add comments explaining the indexing for future me

__global__ void matmul_kernel(float *A, float *B, float *C, int M, int N, int K) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row >= M || col >= N) return;

  float sum = 0;
  for (int i = 0; i < K; ++i) {
    sum += A[row * K + i] * B[col + i * N]; 
  }

  C[row * N + col] = sum;
}

__global__ void matmul_kernel_b_transpose(float *A, float *B, float *C, int M, int N, int K) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row >= M || col >= N) return;

  float sum = 0;
  for (int i = 0; i < K; ++i) {
    sum += A[row * K + i] * B[col * K + i]; 
  }

  C[row * N + col] = sum;
}

void matmul_cpu(float *A, float *B, float *C, int M, int N, int K) {
  for (int r = 0; r < M; ++r) {
    for (int c = 0; c < N; ++c) {
      float sum = 0;
      for (int i = 0; i < K; ++i) {
        sum += A[r * K + i] * B[c + i * N];
      }
      C[r * N + c] = sum;
    }
  }
}

int main() {
  int size_a = M * N;
  int size_b = K * N;
  int size_c = M * N;

  // allocate memory on host
  float *A_h = rand_init(size_a);
  float *B_h = rand_init(size_b);
  float *C_h = malloc(size_c * sizeof(float));

  // allocate memory on device
  float *A_d, *B_d, *C_d;
  check_err(hipMalloc(&A_d, size_a * sizeof(float)));
  check_err(hipMalloc(&B_d, size_b * sizeof(float)));
  check_err(hipMalloc(&C_d, size_c * sizeof(float)));

  // move the A and B matrice to device
  check_err(hipMemcpy(A_d, A_h, size_a * sizeof(float), hipMemcpyHostToDevice));
  check_err(hipMemcpy(B_d, B_h, size_b * sizeof(float), hipMemcpyHostToDevice));

  // launch the kernel
  dim3 block(16, 16);
  dim3 grid(ceil(N / 16.0), ceil(M / 16.0));
  matmul_kernel<<<grid, block>>>(A_d, B_d, C_d, M, N);

  // TODO: implement CPU matmul for correctness testing

  // copy result to host
  check_err(hipMemcpy(C_h, C_d, size_c * sizeof(float), hipMemcpyDeviceToHost));

  // TODO: compare against the CPUs matmul computation

  return 0;
}
