#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include "utils.h"

void vec_add_cpu(float *A_h, float *B_h, float *C_h, int n) {
  for (int i = 0; i < n; ++i)
    C_h[i] = A_h[i] + B_h[i];
}

__global__
void vec_add_kernel(float *A, float *B, float *C, int n) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < n) {
    C[i] = A[i] + B[i];
  }
}

void vec_add(float *A_h, float* B_h, float* C_h, int n) {
  int size = n * sizeof(float);
  float *A_d, *B_d, *C_d;

  // Part 1: Allocate device memory for A, B and C
  check_err(hipMalloc(&A_d, size));
  check_err(hipMalloc(&B_d, size));
  check_err(hipMalloc(&C_d, size));
  
  // Copy A and B to device memory
  // (dest, source, size, direction)
  check_err(hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice));
  check_err(hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice));
  
  // Part 2: call kernel - to launch a grid of threads
  // to perform the actual vector addition
  vec_add_kernel<<<ceil(n/256.0), 256>>>(A_d, B_d, C_d, size);

  // Part 3: Copy C from this device memory
  check_err(hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost));

  // Free device vectors
  check_err(hipFree(A_d));
  check_err(hipFree(B_d));
  check_err(hipFree(C_d));
}

int main() {
  // set the seed for the random number generator
  srand(time(NULL));

  // allocate memory for the vectors
  int N = 10;

  float *A_h = rand_init(N);
  float *B_h = rand_init(N);
  float *C_h = (float*) malloc(N * sizeof(float));

  vec_add(A_h, B_h, C_h, N);

  print_arr(A_h, N);
  print_arr(B_h, N);
  print_arr(C_h, N);

  // free allocated memory
  free(A_h);
  free(B_h);
  free(C_h);
}
