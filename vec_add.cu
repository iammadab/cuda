#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include "utils.h"

__global__
void vec_add_kernel(float *A, float *B, float *C, int n) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < n) {
    C[i] = A[i] + B[i];
  }
}

void vec_add(float *A_h, float* B_h, float* C_h, int n) {
  int size = n * sizeof(float);
  float *A_d, *B_d, *C_d;

  // Part 1: Allocate device memory for A, B and C
  check_err(hipMalloc(&A_d, size));
  check_err(hipMalloc(&B_d, size));
  check_err(hipMalloc(&C_d, size));
  
  // Copy A and B to device memory
  // (dest, source, size, direction)
  check_err(hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice));
  check_err(hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice));

  // create cuda events for timing
  hipEvent_t start, stop;
  check_err(hipEventCreate(&start));
  check_err(hipEventCreate(&stop));

  int grid_size = ceil(n/256.0);
  int block_size = 256;

  // warm up
  for (int i = 0; i < 10; i++) {
    vec_add_kernel<<<grid_size, block_size>>>(A_d, B_d, C_d, size);
  }
  hipDeviceSynchronize();

  int runs = 100;
  float total_ms = 0;

  for (int i = 0; i < runs; i++) {
    // Record the start event
    check_err(hipEventRecord(start));
    
    // Part 2: call kernel - to launch a grid of threads
    // to perform the actual vector addition
    vec_add_kernel<<<ceil(n/256.0), 256>>>(A_d, B_d, C_d, size);

    // Record the stop event
    check_err(hipEventRecord(stop));
    check_err(hipEventSynchronize(stop));

    // calculate the elapsed time
    float ms = 0;
    check_err(hipEventElapsedTime(&ms, start, stop));
    total_ms += ms;
  }

  printf("Average kernel execution time: %f ms \n", total_ms / runs);

  // Part 3: Copy C from this device memory
  check_err(hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost));

  // Free device vectors
  check_err(hipFree(A_d));
  check_err(hipFree(B_d));
  check_err(hipFree(C_d));
}

int main() {
  // set the seed for the random number generator
  srand(time(NULL));

  // allocate memory for the vectors
  int N = 1000000;

  float *A_h = rand_init(N);
  float *B_h = rand_init(N);
  float *C_h = (float*) malloc(N * sizeof(float));

  vec_add(A_h, B_h, C_h, N);

  // free allocated memory
  free(A_h);
  free(B_h);
  free(C_h);
}
