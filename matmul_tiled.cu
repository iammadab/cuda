#include "hip/hip_runtime.h"
#define UTILS_IMPLEMENTATION
#include "utils.h"

int M = 320;
int K = 320;
int N = 320;

// MATMUL KERNEL
// C = A x B
//
// Dimensions (row, col)
// A = (M, K)
// B = (K, N)
// C = (M, N)

// TODO: plot improvement with different tile widths

#define TILE_WIDTH 16

__global__ void matmul_kernel_tiled(float *A, float *B, float *C, int M, int N, int K) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ float Ads[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Bds[TILE_WIDTH][TILE_WIDTH];

  float sum = 0;

  for (int phase = 0; phase < (K + TILE_WIDTH - 1) / TILE_WIDTH; ++phase) {
    int ph_row = phase * TILE_WIDTH + threadIdx.y;
    int ph_col = phase * TILE_WIDTH + threadIdx.x;

    if (row < M && ph_col < K)
      Ads[threadIdx.y][threadIdx.x] = A[row * K + ph_col];
    else
      Ads[threadIdx.y][threadIdx.x] = 0.0f;

    if (ph_row < K && col < N)
      Bds[threadIdx.y][threadIdx.x] = B[ph_row * N + col];
    else
      Bds[threadIdx.y][threadIdx.x] = 0.0f;

    __syncthreads();
  
    for (int i = 0; i < TILE_WIDTH; ++i) {
      sum += Ads[threadIdx.y][i] * Bds[i][threadIdx.x];
    }
    __syncthreads();
  }

  if (row < M && col < N)
    C[row * N + col] = sum;
}

int main() {
  int size_a = M * K;
  int size_b = K * N;
  int size_c = M * N;

  // allocate memory on host
  float *A_h = rand_init(size_a);
  float *B_h = rand_init(size_b);
  float *C_h = (float *) malloc(size_c * sizeof(float));
  float *C_h_cpu_result = (float *) malloc(size_c * sizeof(float));

  // allocate memory on device
  float *A_d, *B_d, *C_d;
  CHECK_ERR(hipMalloc(&A_d, size_a * sizeof(float)));
  CHECK_ERR(hipMalloc(&B_d, size_b * sizeof(float)));
  CHECK_ERR(hipMalloc(&C_d, size_c * sizeof(float)));


  // copy data to host
  CHECK_ERR(hipMemcpy(A_d, A_h, size_a * sizeof(float), hipMemcpyHostToDevice));
  CHECK_ERR(hipMemcpy(B_d, B_h, size_b * sizeof(float), hipMemcpyHostToDevice));

  // compute expected answer on the cpu
  matmul_cpu(A_h, B_h, C_h_cpu_result, M, N, K);

  // kernel parameters
  dim3 block(TILE_WIDTH, TILE_WIDTH);
  dim3 grid(ceil(N / (float) TILE_WIDTH), ceil(M / (float) TILE_WIDTH));

  // comparison parameters
  const int WARMUP_COUNT = 3;
  const int REPEAT_COUNT = 10;
  float eps = 1e-4f;

  hipEvent_t start, stop;
  CHECK_ERR(hipEventCreate(&start));
  CHECK_ERR(hipEventCreate(&stop));

  // tiled matmul kernel
  for (int i = 0; i < WARMUP_COUNT; ++i) {
    matmul_kernel_tiled<<<grid, block>>>(A_d, B_d, C_d, M, N, K);
  }
  CHECK_ERR(hipDeviceSynchronize());

  // timed run
  CHECK_ERR(hipEventRecord(start));
  for (int i = 0; i < REPEAT_COUNT; ++i) {
    matmul_kernel_tiled<<<grid, block>>>(A_d, B_d, C_d, M, N, K);
  }
  CHECK_ERR(hipEventRecord(stop));
  CHECK_ERR(hipEventSynchronize(stop));

  // copy result to host
  CHECK_ERR(hipMemcpy(C_h, C_d, size_c * sizeof(float), hipMemcpyDeviceToHost));
  hipDeviceSynchronize();

  for (int i = 0; i < size_c; ++i) {
    if (fabsf(C_h_cpu_result[i] - C_h[i]) > eps) {
      fprintf(stderr, "result mismatch");
      return 1;
    }
  }

  float ms = 0;
  CHECK_ERR(hipEventElapsedTime(&ms, start, stop));
  ms /= REPEAT_COUNT;

  printf("ok tiled matmul: %fms\n", ms);

  // free memory
  free(A_h);
  free(B_h);
  free(C_h);
  free(C_h_cpu_result);
  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);

  return 0;
}
