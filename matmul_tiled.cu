#include "hip/hip_runtime.h"
#define UTILS_IMPLEMENTATION
#include "utils.h"

int M = 100;
int K = 100;
int N = 100;

// MATMUL KERNEL
// C = A x B
//
// Dimensions (row, col)
// A = (M, K)
// B = (K, N)
// C = (M, N)

// TODO: plot improvement with different tile widths

#define TILE_WIDTH 16

// TODO: handle irregular sized tile_widths

__global__ void matmul_kernel_tiled(float *A, float *B, float *C, int K) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ float Ads[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Bds[TILE_WIDTH][TILE_WIDTH];

  float sum = 0;

  for (int phase = 0; phase < K / TILE_WIDTH; ++phase) {
    Ads[row][col] = A[row * K + (phase * TILE_WIDTH + threadIdx.x)];
    Bds[row][col] = B[(phase * TILE_WIDTH + threadIdx.y) * K + col];
    __syncthreads();

    for (int i = 0; i < TILE_WIDTH; ++i) {
      sum += Ads[row][i] * Bds[i][col];
    }
    __syncthreads();
  }

  C[row * K + col] = sum;
}

int main() {
  int size_a = M * K;
  int size_b = K * N;
  int size_c = M * N;

  // allocate memory on host
  float *A_h = rand_init(size_a);
  float *B_h = rand_init(size_b);
  float *C_h = (float *) malloc(size_c * sizeof(float));
  float *C_h_cpu_result = (float *) malloc(size_c * sizeof(float));

  // allocate memory on device
  float *A_d, *B_d, *C_d;
  check_err(hipMalloc(&A_d, size_a * sizeof(float)));
  check_err(hipMalloc(&B_d, size_b * sizeof(float)));
  check_err(hipMalloc(&C_d, size_c * sizeof(float)));

  // copy data to host
  check_err(hipMemcpy(A_d, A_h, size_a * sizeof(float), hipMemcpyDeviceToHost));
  check_err(hipMemcpy(B_d, B_h, size_b * sizeof(float), hipMemcpyDeviceToHost));

  // compute expected answer on the cpu
  matmul_cpu(A_h, B_h, C_h_cpu_result, M, N, K);

  // kernel parameters
  dim3 block(TILE_WIDTH, TILE_WIDTH);
  dim3 grid(ceil(N / (float) TILE_WIDTH), ceil(M / (float) TILE_WIDTH));

  // TODO: use the tile_width as the block size
  // TODO: compute the GRID size accordingly

  return 0;
}
