#include "hip/hip_runtime.h"
#define UTILS_IMPLEMENTATION
#include "utils.h"

int M = 100;
int K = 100;
int N = 100;

// MATMUL KERNEL
// C = A x B
//
// Dimensions (row, col)
// A = (M, K)
// B = (K, N)
// C = (M, N)

// TODO: plot improvement with different tile widths

#define TILE_WIDTH 16

// TODO: handle irregular sized tile_widths

__global__ void matmul_kernel_tiled(float *A, float *B, float *C, int K) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ float Ads[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Bds[TILE_WIDTH][TILE_WIDTH];

  float sum = 0;

  for (int phase = 0; phase < K / TILE_WIDTH; ++phase) {
    Ads[row][col] = A[row * K + (phase * TILE_WIDTH + threadIdx.x)];
    Bds[row][col] = B[(phase * TILE_WIDTH + threadIdx.y) * K + col];
    __syncthreads();

    for (int i = 0; i < TILE_WIDTH; ++i) {
      sum += Ads[row][i] * Bds[i][col];
    }
    __syncthreads();
  }

  C[row * K + col] = sum;
}

int main() {
  int size_a = M * K;
  int size_b = K * N;
  int size_c = M * N;

  // allocate memory on host
  float *A_h = rand_init(size_a);
  float *B_h = rand_init(size_b);
  float *C_h = (float *) malloc(size_c * sizeof(float));
  float *C_h_cpu_result = (float *) malloc(size_c * sizeof(float));

  // allocate memory on device
  float *A_d, *B_d, *C_d;
  check_err(hipMalloc(&A_d, size_a * sizeof(float)));
  check_err(hipMalloc(&B_d, size_b * sizeof(float)));
  check_err(hipMalloc(&C_d, size_c * sizeof(float)));

  // copy data to host
  check_err(hipMemcpy(A_d, A_h, size_a * sizeof(float), hipMemcpyDeviceToHost));
  check_err(hipMemcpy(B_d, B_h, size_b * sizeof(float), hipMemcpyDeviceToHost));

  // compute expected answer on the cpu
  matmul_cpu(A_h, B_h, C_h_cpu_result, M, N, K);

  // kernel parameters
  dim3 block(TILE_WIDTH, TILE_WIDTH);
  dim3 grid(ceil(N / (float) TILE_WIDTH), ceil(M / (float) TILE_WIDTH));

  // comparison parameters
  const int WARMUP_COUNT = 3;
  const int REPEAT_COUNT = 10;
  float eps = 1e-4f;

  hipEvent_t start, stop;
  check_err(hipEventCreate(&start));
  check_err(hipEventCreate(&start));

  // tiled matmul kernel
  for (int i = 0; i < WARMUP_COUNT; ++i) {
    matmul_kernel_tiled<<<grid, block>>>(A_d, B_d, C_d, K);
  }
  check_err(hipDeviceSynchronize());

  // timed run
  check_err(hipEventRecord(start));
  for (int i = 0; i < REPEAT_COUNT; ++i) {
    matmul_kernel_tiled<<<grid, block>>>(A_d, B_d, C_d, K);
  }
  check_err(hipEventRecord(stop));
  check_err(hipEventSynchronize(stop));

  // copy result to host
  check_err(hipMemcpy(C_h, C_d, size_c * sizeof(float), hipMemcpyDeviceToHost));
  hipDeviceSynchronize();

  for (int i = 0; i < size_c; ++i) {
    if (fabsf(C_h_cpu_result[i] - C_h[i]) > eps) {
      fprintf(stderr, "result mismatch");
      return 1;
    }
  }

  float ms = 0;
  check_err(hipEventElapsedTime(&ms, start, stop));
  ms /= REPEAT_COUNT;

  printf("ok tiled matmul: %fms\n", ms);

  // free memory
  free(A_h);
  free(B_h);
  free(C_h);
  free(C_h_cpu_result);
  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);

  return 0;
}
